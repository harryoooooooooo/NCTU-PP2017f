
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdbool.h>

typedef unsigned long long int ull;

__device__ bool getval(int v, ull id, ull ie){
    if (v<0) v=-v;
    if (v<=30) return (id & (1llu<<v)) ? true : false;
    return (ie & (1llu<<(v-31))) ? true : false;
}

__device__ bool test(int n, int* raw, ull id, ull ie){
    bool ret = true;
    for (int i = 0; i < n; i+=3){
        bool tmp = false;
        for (int j = 0; j < 3; j++)
            tmp |= (getval(raw[i+j], id, ie) ^ (raw[i+j] < 0));
        ret &= tmp;
    }
    return ret;
}

__device__ void fillres(int m, bool* res, ull id, ull ie){
    for (int i=1;i<=m;i++)
        res[i] = getval(i, id, ie);
}

__global__ void bf(int n, int m, int* raw, bool* res, int* flag){
    ull myid = blockIdx.x * 1024llu + threadIdx.x;
    ull mxstate = (1llu<<m) - 1;
    if (myid > mxstate) return;
    ull end = 1;
    if (m-30 > 0) end <<= m-30;
    for (ull i = 0; i < end; i ++){
        if (test(n, raw, myid<<1, i)){
            if (!atomicExch(flag, 1))
                fillres(m, res, myid<<1, i);
            return;
        }
        if ((i & 0xff) == (myid & 0xff) && *flag)
            return;
    }
}

int main (){
    int *rawd, *raw, *flag;
    bool *resd, *res;
    int n, m, mflag = 0;
    scanf("%d%d", &n,&m);
    n*=3;
    raw = (int*)malloc(sizeof(int)*n);
    res = (bool*)malloc(m+1);
    for (int i=0;i<n;i++)
        scanf("%d", raw+i);
    hipMalloc((void**)&rawd, sizeof(int)*n);
    hipMalloc((void**)&resd, m+1);
    hipMalloc((void**)&flag, sizeof(int));
    hipMemcpy(rawd, raw, sizeof(int)*n, hipMemcpyHostToDevice);
    hipMemcpy(flag, &mflag, sizeof(int), hipMemcpyHostToDevice);
    bf<<<1048576,1024>>>(n, m, rawd, resd, flag);
    hipMemcpy(&mflag, flag, sizeof(int), hipMemcpyDeviceToHost);
    if (mflag){
        hipMemcpy(res, resd, m+1, hipMemcpyDeviceToHost);
        for (int i = 1; i <= m; i++)
            printf("%d := %s\n", i, (res[i]?"true":"false"));
    }
    else printf("No satisfy!\n");
    hipFree(rawd);
    hipFree(resd);
    hipFree(flag);
    free(raw);
    free(res);
}
